#include "hip/hip_runtime.h"
#include "knn.cuh"
#include <stdio.h>
#include <iostream>
#define BLOCK_DIM 16
using namespace std;

__global__ void calDistance_kernel(float *A, float *B, float *dist, unsigned int nA, unsigned int nB, unsigned int dim) {
    __shared__ float sMem_a[BLOCK_DIM][BLOCK_DIM];
    __shared__ float sMem_b[BLOCK_DIM][BLOCK_DIM];

    // rename threadidx
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum = 0.f; 

    // setting start and end range
    // checking matrix A by y, and matrix B by x, because we will consider cross multiply
    // __shared__ int Bg_a = BLOCK_DIM * blockIdx.y;
    // __shared__ int Sp_a = BLOCK_DIM * nA;           // moving begin pointer down the matrix
    // __shared__ int Ed_a = Bg_a + nA * (dim-1);      // end at the last element in the matrix
    // __shared__ int Bg_b = BLOCK_DIM * blockIdx.x;
    // __shared__ int Ed_b = BLOCK_DIM * nB;

    __shared__ int Bg_a, Sp_a, Ed_a, Bg_b, Sp_b;
    Bg_a = BLOCK_DIM * blockIdx.y;
    Sp_a = BLOCK_DIM * nA; 
    Ed_a = Bg_a + nA * (dim-1);
    Bg_b = BLOCK_DIM * blockIdx.x;
    Sp_b = BLOCK_DIM * nB;

    for (int i=Bg_a, j=Bg_b; i<Ed_a; i+=Sp_a, j+=Sp_b) {
        // add element value to share memory, if index i is out-of-bound, fill shared memory with 0
        // if (tx==0 && ty ==0) {
        //   printf("dim is %d, ", dim);
        //   printf("i/nA + ty is %d, ", i/nA + ty);
        //   printf("Bg_a + tx is %d, ", Bg_a + tx);
        //   printf("A value is %f", A[i + ty*nA + tx]);
        //   printf("here");
        // }
        if (i/nA + ty < dim) {
            if (Bg_a + tx < nA) {
                sMem_a[ty][tx] = A[i + ty*nA + tx];
            } else {
                sMem_a[ty][tx] = 0;
            }

            // if (tx==0 && ty ==0) {
            //   printf("%.2f\n", sMem_a[ty][tx]);
            //   // should be 1 here
            // }

            // if (tx==0 && ty ==1) {
            //   printf("%.2f\n", sMem_a[ty][tx]);
            //   // should be 2 here
            // }

            if (Bg_b + tx < nB) {
                sMem_b[ty][tx] = B[j + ty*nB + tx];
            } else {
                sMem_b[ty][tx] = 0;
            }

            // if (tx==0 && ty ==0) {
            //   printf("%.2f\n", sMem_b[ty][tx]);
            //   // should be 4 here
            // }

            // if (tx==0 && ty ==1) {
            //   printf("%.2f\n", sMem_b[ty][tx]);
            //   // should be 5 here
            // }
        } else {
            sMem_a[ty][tx] = 0;
            sMem_b[ty][tx] = 0;
        }
        __syncthreads();

        // calculate distant follows format: dist = (x_A - x_B)^2
        if (Bg_b + tx < nB && Bg_a + ty < nA) {
            for (int k=0; k<BLOCK_DIM; k++) {
                sum += pow(sMem_a[k][ty] - sMem_b[k][tx], 2);
                // if (tx==0 && ty ==0) {
                //   printf("%.2f\n", sum);
                // }
            }
        }
        __syncthreads();
    }

    // store dist in dist matrix
    if (Bg_b + tx < nB && Bg_a + ty < nA) {
        // if (tx==0 && ty ==0) {
        //   printf("%.2f\n", sum);
        // }
        dist[(Bg_a+ty)*nB + Bg_b+tx] = sum;
        // if (tx==3 && ty == 9) {
        //   printf("dist: %.2f\n", dist[0]);
        // }   
    }
}


// may had to rewrite sort
__global__ void sortDistance_kernel(float *dist, int *idx, unsigned int w, unsigned int h, unsigned int k) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < w) {
        float* dist_col = dist + tid;
        int* idx_col = idx + tid;

        int l = 0, r = h-1;
        // find the kth smallest element in dist and return their corresponding idx with quicksort
        while (l < r) {
            int pivot = dist_col[r * w];
            int i = l-1;

            for (int j=l; j<= r-1; j++) {
                if (dist_col[j*w] <= pivot) {
                    i++;
                    // swap dist_col[i] and dist_col[j]
                    int temp = dist_col[i*w];
                    dist_col[i*w] = dist_col[j*w];
                    dist_col[j*w] = temp;

                    // swap there corresponding indices
                    int temp_idx = idx_col[i*w];
                    idx_col[i*w] = idx_col[j*w];
                    idx_col[j*w] = temp_idx;
                }
            }

            int temp = dist_col[(i+1)*w];
            dist_col[(i+1)*w] = dist_col[r*w];
            dist_col[r*w] = temp;

            // swap corresponding indices
            int temp_idx = idx_col[(i+1)*w];
            idx_col[(i+1)*w] = idx_col[r*w];
            idx_col[r*w] = temp_idx;

            int partition_idx = i+1;

            if (partition_idx == k) break;

            if (partition_idx > k) r = partition_idx -1;
            else l = partition_idx -1;
            
        }
    }
}

void KNN(float *A, 
         unsigned int nA, 
         float *B, 
         unsigned int nB, 
         unsigned int dim, 
         unsigned int k, 
         float *dist, 
         int *idx) {
    
    dim3 blk_dim_cal((nB+BLOCK_DIM-1)/BLOCK_DIM, (nA+BLOCK_DIM-1)/BLOCK_DIM, 1);
    dim3 thrd_per_blk_cal(BLOCK_DIM, BLOCK_DIM, 1);

    int thrd_per_blk_sort = BLOCK_DIM*BLOCK_DIM;
    int blk_dim_sort = (nB + thrd_per_blk_sort -1) / thrd_per_blk_sort;

    calDistance_kernel<<<blk_dim_cal, thrd_per_blk_cal>>>(A, B, dist, nA, nB, dim);
    
    // cout << "Distance Matrix in KNN.CU:" << endl;
    // for (unsigned int i=0; i<20; i++) {
    //   cout << dist[i] << " ";
    // }
    // there is no output may be attribute to asyn in cpu & gpu

    sortDistance_kernel<<<blk_dim_sort, thrd_per_blk_sort>>>(dist, idx, nB, nA, k);

    hipDeviceSynchronize();

    for (int i = 0; i < nA; ++i) {
        for (int j = 0; j < nB; ++j) {
            cout << dist[i*nB + j] << " ";
        }
        cout << endl;
    }

    cout << "below is index matrix" << endl;
    for (int i = 0; i < nA; ++i) {
        for (int j = 0; j < nB; ++j) {
            cout << idx[i*nB + j] << " ";
        }
        cout << endl;
    }
    
}
